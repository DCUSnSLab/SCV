#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cupti.h>
#include <stdint.h>
#include <map>

#define CUPTI_CALL(call)                                                    \
    do {                                                                    \
        CUptiResult _status = call;                                         \
        if (_status != CUPTI_SUCCESS) {                                     \
            const char *errstr;                                             \
            cuptiGetResultString(_status, &errstr);                         \
            fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",\
                    __FILE__, __LINE__, #call, errstr);                     \
            exit(-1);                                                       \
        }                                                                   \
    } while (0)

#define CUDA_CALL(call)                                                     \
    do {                                                                    \
        hipError_t err = call;                                             \
        if (err != hipSuccess) {                                           \
            fprintf(stderr, "%s:%d: CUDA error: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(-1);                                                       \
        }                                                                   \
    } while (0)

// Dummy kernel to generate GPU activity
__global__ void dummyKernel(float *data, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        data[idx] = data[idx] * 2.0f;
    }
}

int main(void) {
    int N = 256;
    float *d_data;
    CUDA_CALL(hipMalloc((void**)&d_data, N * sizeof(float)));
    dummyKernel<<<(N + 63) / 64, 64>>>(d_data, N);
    CUDA_CALL(hipDeviceSynchronize());

    CUptiResult status;
    size_t bufferSize = 1024 * 1024;  // Allocate 1MB buffer
    uint8_t *buffer = (uint8_t *) malloc(bufferSize);
    if (buffer == NULL) {
        fprintf(stderr, "Error: Memory allocation failed\n");
        exit(-1);
    }

    // Enable CUPTI activities: enable both context and kernel activities.
    CUPTI_CALL(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_CONTEXT));
    CUPTI_CALL(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_KERNEL));

    // Synchronize the device to trigger GPU activities.
    CUDA_CALL(hipDeviceSynchronize());

    // Flush any remaining records from CUPTI's internal buffer.
    CUPTI_CALL(cuptiActivityFlushAll(0));

    // Map to associate context ID with PID.
    std::map<uint32_t, uint32_t> contextPidMap;
    // Map to accumulate kernel execution time (in nanoseconds) per PID.
    std::map<uint32_t, uint64_t> pidKernelTime;

    CUpti_Activity *record = NULL;
    while ((status = cuptiActivityGetNextRecord(buffer, bufferSize, &record)) == CUPTI_SUCCESS) {
        switch (record->kind) {
            case CUPTI_ACTIVITY_KIND_CONTEXT: {
                CUpti_ActivityContext *ctx = (CUpti_ActivityContext *) record;
                uint32_t processId = 0;
                size_t attrSize = sizeof(processId);
#ifdef CUPTI_ACTIVITY_ATTR_CONTEXT_PROCESS_ID
                // If the CUPTI version supports the process id attribute:
                CUPTI_CALL(cuptiActivityGetObjectAttribute(
                    ctx->contextId,
                    CUPTI_ACTIVITY_OBJECT_CONTEXT,
                    CUPTI_ACTIVITY_ATTR_CONTEXT_PROCESS_ID,
                    &attrSize,
                    &processId));
#else
                // If the CUPTI version does not support the process id attribute,
                // set processId to 0.
                processId = 0;
#endif
                contextPidMap[ctx->contextId] = processId;
                break;
            }
            case CUPTI_ACTIVITY_KIND_KERNEL: {
                // Cast kernel record to CUpti_ActivityKernel4 (latest CUPTI).
                CUpti_ActivityKernel4 *kernel = (CUpti_ActivityKernel4 *) record;
                uint32_t pid = 0;
                if (contextPidMap.find(kernel->contextId) != contextPidMap.end()) {
                    pid = contextPidMap[kernel->contextId];
                }
                // Kernel execution time is provided in nanoseconds.
                uint64_t duration = kernel->end - kernel->start;
                pidKernelTime[pid] += duration;
                break;
            }
            default:
                // Ignore other types of records.
                break;
        }
    }

    if (status != CUPTI_ERROR_MAX_LIMIT_REACHED) {
        const char *errstr;
        cuptiGetResultString(status, &errstr);
        fprintf(stderr, "cuptiActivityGetNextRecord failed: %s\n", errstr);
    }

    // Print the GPU kernel execution time per PID in milliseconds.
    printf("GPU kernel execution time per PID (in milliseconds):\n");
    for (auto it = pidKernelTime.begin(); it != pidKernelTime.end(); ++it) {
        double ms = (double) it->second / 1e6; // Convert nanoseconds to milliseconds
        printf("PID %u: %.3f ms\n", it->first, ms);
    }

    free(buffer);

    // Disable CUPTI activities.
    CUPTI_CALL(cuptiActivityDisable(CUPTI_ACTIVITY_KIND_CONTEXT));
    CUPTI_CALL(cuptiActivityDisable(CUPTI_ACTIVITY_KIND_KERNEL));

    CUDA_CALL(hipFree(d_data));
    return 0;
}
