#include "hip/hip_runtime.h"
#include <ros/ros.h>
#include <std_msgs/String.h>
#include <hip/hip_runtime.h>

// CUDA 커널: 반복적인 행렬 덧셈 연산을 수행하여 GPU 부하 증가
__global__ void heavyKernel(float* d_A, float* d_B, float* d_C, int N, int iterations) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        for (int i = 0; i < iterations; i++) {  // GPU 연산 반복
            d_C[idx] = d_A[idx] + d_B[idx];
        }
    }
}

void gpuComputationLoop() {
    const int N = 1024 * 1024 *10;  // 기존보다 100배 증가 (100MB 데이터)
    const int iterations = 10000;  // 연산 반복 횟수 증가
    float *d_A, *d_B, *d_C;

    // GPU 메모리 할당 (CPU 메모리 할당 X)
    hipMalloc((void**)&d_A, N * sizeof(float));
    hipMalloc((void**)&d_B, N * sizeof(float));
    hipMalloc((void**)&d_C, N * sizeof(float));

    // 데이터 초기화 (GPU에서 직접 수행)
    hipMemset(d_A, 1, N * sizeof(float));
    hipMemset(d_B, 2, N * sizeof(float));

    // CUDA 커널 실행 설정
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);

    while (ros::ok()) {
        heavyKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, N, iterations);
        hipDeviceSynchronize();  // 연산 완료 대기

        ROS_INFO("Heavy CUDA kernel executed successfully");

        ros::Duration(1.0).sleep();  // 1초 대기
    }

    // GPU 메모리 해제
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main(int argc, char** argv) {
    ros::init(argc, argv, "dummy_cuda_node");
    ros::NodeHandle nh;

    ROS_INFO("Dummy CUDA Node Started");

    gpuComputationLoop();  // GPU를 계속 사용하는 연산 실행

    return 0;
}
